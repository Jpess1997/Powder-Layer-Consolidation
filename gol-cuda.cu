#include "hip/hip_runtime.h"
//Secondary function code to run on the GPU with cuda
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <stdlib.h>
#include <math.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

float nnodesl;

extern int *ID;

extern float *coordinates;

extern float *elements;

extern int nel, nnodes, np, nzmax;

extern char baseName[80];

extern float *psi;

extern float *LM;

extern int *irow;

extern int *icol;

extern float powderThick;

float *N;

float *dN;

float jac;

float *ke;

float *fe;

extern float *d;

extern float *a_bar;

extern "C"
{
  extern void num_ElementsNodes(char baseName [80], int myrank, int nnodes);
  
  extern void read_coordinates(char baseName [80], int myrank, int nnodes);

  extern void read_elements(char baseName [80], int myrank, int nel);

  extern void read_psi(char baseName [80], int myrank, int nel);

  extern bool gol_runKernel(float *coordinates, int nnodes, float powderThick,
			    float Tol, float *elements,
			    int nel, int **d_ID, ushort threadsCount,
			    float **d_d, float **d_a_bar);

  extern void gol_freeData();
}

//This is an debugging function that prints out the most recent line the code gets to when running before an error occurs
void printLine(int line)
{
  char fileName[30]; //initializes file buffer
  snprintf(fileName,100,"errorAtLineCuda.txt"); //created file name to be sent to
  FILE *fp;
  fp = fopen(fileName,"w+"); //opens file
  fprintf(fp,"Line is %d.\n",line); //prints the current line number to the file
  fclose(fp); //closes file
}

//This funciton takes in the number of nodes from the user and produces the number of elemements
void num_ElementsNodes(char baseName [80], int myrank, int nnodes)
{
  nel = nnodes * 6; //generates number of elements for specified number of nodes
}

//Generates the dummy coordinate values based off of the specified nodes
void read_coordinates(char baseName [80], int myrank, int nnodes)
{
  int a,b;
  printLine(__LINE__);
  hipMallocManaged(&coordinates, ((nnodes * 3) * sizeof(float))); //allocates memory
  for (a=0; a<(nnodes); a++)
    {
      for(b=0;b<3;b++)
	{
	  coordinates[a+b*nnodes] = 0; //assigns each coordinate location as zero
	}
    }
}

//Generates dummy element values based off of number of elements
void read_elements(char baseName [80], int myrank, int nel)
{
  hipMallocManaged(&elements, ((nel * 4) * sizeof(float)));
  int a;
  for (a=0; a<(4*nel); a++)
    {
      elements[a] = 0; //assigns each element value as zero
    }
}

//Generates dummy psi values associated with each element
void read_psi(char baseName [80], int myrank, int nel)
{
  hipMallocManaged(&psi, (nel * sizeof(float)));
  int a;
  for(a=0;a<nel;a++)
    {
      psi[a] = 0.5; //loads value for psi paramater to each element
    }
}

//Shape function for the FEM approach to deform the mesh
void shape(float gp[3], float xe[12])
{
  int i;
  //local coordinate
  float r = gp[0];
  float s = gp[1];
  float t = gp[2];

  //Shape functions
  N[0] = r;
  N[1] = s;
  N[2] = t;
  N[3] = 1-r-s-t;

  //shape functions at node points
  float N_r[4] = {1, 0, 0, -1};
  float N_s[4] = {0, 1, 0, -1};
  float N_t[4] = {0 ,0 , 1, -1};

  //xyz locations of shape functions at node points
  float x_r = N_r[0]*xe[0+0*4] + N_r[1]*xe[1+0*4] + N_r[2]*xe[2+0*4] + N_r[3]*xe[3+0*4];
  float x_s = N_s[0]*xe[0+0*4] + N_s[1]*xe[1+0*4] + N_s[2]*xe[2+0*4] + N_s[3]*xe[3+0*4];
  float x_t = N_t[0]*xe[0+0*4] + N_t[1]*xe[1+0*4] + N_t[2]*xe[2+0*4] + N_t[3]*xe[3+0*4];

  float y_r = N_r[0]*xe[0+1*4] + N_r[1]*xe[1+1*4] + N_r[2]*xe[2+1*4] + N_r[3]*xe[3+1*4];
  float y_s = N_s[0]*xe[0+1*4] + N_s[1]*xe[1+1*4] + N_s[2]*xe[2+1*4] + N_s[3]*xe[3+1*4];
  float y_t = N_t[0]*xe[0+1*4] + N_t[1]*xe[1+1*4] + N_t[2]*xe[2+1*4] + N_t[3]*xe[3+1*4];

  float z_r = N_r[0]*xe[0+2*4] + N_r[1]*xe[1+2*4] + N_r[2]*xe[2+2*4] + N_r[3]*xe[3+2*4];
  float z_s = N_s[0]*xe[0+2*4] + N_s[1]*xe[1+2*4] + N_s[2]*xe[2+2*4] + N_s[3]*xe[3+2*4];
  float z_t = N_t[0]*xe[0+2*4] + N_t[1]*xe[1+2*4] + N_t[2]*xe[2+2*4] + N_t[3]*xe[3+2*4];

  //determinant of the jacobian
  float jacDet = x_r*(y_s*z_t - y_t*z_s) - x_s*(y_r*z_t - y_t*z_r) + x_t*(y_r*z_s - y_s*z_r);

  //jacobian
  jac = abs(jacDet);

  //inverse of the jacobian
  float inv_jac[3][3] = {(y_s*z_t - y_t*z_s)/jacDet, (x_t*z_s - x_s*z_t)/jacDet, (x_s*y_t - x_t*y_s)/jacDet,
                         (y_t*z_r - y_r*z_t)/jacDet, (x_r*z_t - x_t*z_r)/jacDet, (x_t*y_r - x_r*y_t)/jacDet,
                         (y_r*z_s - y_s*z_r)/jacDet, (x_s*z_r - x_r*z_s)/jacDet, (x_r*y_s - x_s*y_r)/jacDet};

  //derivative of the shape function
  for(i=0;i<4;i++)
    {
      dN[i+0*4] = N_r[i]*inv_jac[0][0] + N_s[i]*inv_jac[1][0] + N_t[i]*inv_jac[2][0];
      dN[i+1*4] = N_r[i]*inv_jac[0][1] + N_s[i]*inv_jac[1][1] + N_t[i]*inv_jac[2][1];
      dN[i+2*4] = N_r[i]*inv_jac[0][2] + N_s[i]*inv_jac[1][2] + N_t[i]*inv_jac[2][2];
    }
}

void weakform(float xe[12], float Psie, float porosity)
{
  int i;
  int j;
  int k;
  int l;

  // 1 point formula - degree of precision 1
  float gp[3] = {0.25, 0.25, 0.25};

  //weights
  int w = 1;

  int ngp = 1;

  for(i=0;i<16;i++)
    {
      ke[i] = 0; //initialize ke values
    }
  
  for(i=0;i<4;i++)
    {
      fe[i] = 0; //initialize fe values
    }

  //stress strain displacement matrix
  float B[4] = {0,0,0,0};
  
  //loop over gauss points
  shape(gp,xe); //use shape function outlined above

  for(i=0;i<ngp;i++)
    {
      float por = porosity;

      //current location in z of the point
      float z = N[0]*xe[0+2*4] + N[1]*xe[1+2*4] + N[2]*xe[2+2*4] + N[3]*xe[3+2*4];
      if( z < powderThick)
        {
          por = 0.0; //porosit of the substrate is zero since it's a solid
        }
      for(j=0;j<4;j++)
        {
          B[j] = dN[j*2*4];
        }
      
      //Transpose of N
      float Ntr[4] = {N[0],N[1],N[2],N[3]};
      
      //fill k
      for(k=0;k<4;k++)
	{
	  for(l=0;l<4;l++)
	    {
	      ke[k+1] = ke[k+l] + Ntr[l] * B[k] * w * jac;
	    }
	}
      
      //fill fe
      for(k=0;k<4;k++)
	{
	  fe[k] = fe[k] - Ntr[k] * ((por * Psie)/(1 - por * (1 - Psie)))*w*jac;
	}
    }
}

//function that conversts the solution of the system of equations to the displacements of the powder layer node points.
__global__ void gen_corrector(int nnodes, int *d_ID, float *d_a_bar, float powderThick, float *d_d)
{
  int i, index;

  //Use CUDA to run the displacment converison calculation
  for(i = blockIdx.x * blockDim.x + threadIdx.x; i < nnodes; i += (blockDim.x * gridDim.x))
    {
      index = d_ID[i];
      if(index != 0)
        {
          d_d[i] = d_a_bar[index];
          if(d_d[i] > powderThick)
            {
              d_d[i] = 0;
            }
        }
    }
}

//runs the majority of the displacment calculation and the CUDA kernel outlined above
bool gol_runKernel(float *coordinates, int nnodes, float powderThick,float Tol, float *elements, int nel, int **d_ID, ushort threadsCount, float **d_d, float **d_a_bar)
{
  printLine(__LINE__);
  //Get boundary nodes
  int count = 0;
  int i;
  int j;
  int *fixnodes;
  float z;

  //set fixnodes for boundary nodes to close to the boundary of the powder to the subsrate
  for(i=0;i<nnodes;i++)
    {
      z = coordinates[i+2*nnodes];
      if(fabs(z - powderThick) < Tol)
        {
          fixnodes[count] = i;
          count++;
        }
    }
  printLine(__LINE__);
  hipMallocManaged(&ID, (nnodes * sizeof(int))); //allocate space
  hipMallocManaged(&d, (nnodes * sizeof(float))); //allocate space
  printLine(__LINE__);
  
  //Assembling ID array
  float ID[nnodes];
  for(i=0;i<nnodes;i++)
    {
      ID[i] = 1;
    }
  printLine(__LINE__);
  int ndispl = sizeof(fixnodes)/sizeof(fixnodes[0]);

  int nd;
  int g;
  printf("ndispl is %d\n",ndispl);
  printLine(__LINE__);

  //Adjust ID array based on the nd values
  for(g=0; g<ndispl; g++)
    {
      nd = g;
      ID[nd] = 0;
    }
  printLine(__LINE__);
  
  //Fill ID array
  count = 0;
  for(j=0;j<nnodes;j++)
    {
      if(ID[j] != 0)
        {
          count++;
          ID[j] = count;
        }
    }
  printLine(__LINE__);
  int ndof = 1;

  //initialize values for the displacement vector
  for(i = 0;i < nnodes;i++)
    {
      //Displacement Vector
      d[i] = 0;
      if(ID[i] > ndof)
        {
          ndof = ID[i];
        }
    }
  printf("ndof is %d\n",ndof);
  printLine(__LINE__);

  //For future work implement solver for solving system of equation M * a_bar = F
  hipMallocManaged(&a_bar, (ndof * sizeof(float))); //allocate space
  printLine(__LINE__);
  
  float F[ndof];
  for(i = 0;i < ndof;i++)
    {
      F[i] = 0; //b value to system of equations
    }
  printLine(__LINE__);
  float a_bar[ndof];

  for(i=0; i<ndof;i++)
  {
    a_bar[i] = F[i]; //set a_bar value
  }
  printLine(__LINE__);
  
  //Change small values to zero
  for(i=0;i<ndof;i++)
    {
      if(a_bar[i] < 0.0000001)
        {
          a_bar[i] = 0;
        }
    }
  printLine(__LINE__);
  
  //Corrector phase to be done in cuda 
  size_t reqBlocksCount2 = ceil(nnodes/threadsCount); //number of blocks count for the LM array
  unsigned int blocksCount2 = (unsigned int)min(65536, (unsigned int)reqBlocksCount2); //Blocks count based on required blocks
  gen_corrector<<<blocksCount2, threadsCount>>>(nnodes, *d_ID, *d_a_bar, powderThick, *d_d);
  hipDeviceSynchronize();
  printLine(__LINE__);
  for(i=0;i<nnodes;i++)
    {
      coordinates[i+2*nnodes] = coordinates[i+2*nnodes] + d[i]; //deform coordinates based on d
    }
  printLine(__LINE__);
  return 0;
}

//free the cuda data that is allocated.
void gol_freeData()
{
 hipFree(coordinates);
 hipFree(elements);
 hipFree(psi);
 hipFree(d);
 hipFree(ID);
 hipFree(a_bar);
}
