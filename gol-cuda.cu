#include "hip/hip_runtime.h"
//Secondary function code to run on the GPU with cuda
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <stdlib.h>
#include <math.h>

extern unsigned char *coordinates;

extern unsigned char *elements;

extern size_t nel, nnodes, np, nzmax;

extern char basename;

extern unsigned char *psi;

extern unsigned char *LM;

extern unsigned char *irow;

extern unsigned char *icol;

void num_ElementsNodes(char basename, int myrank)
{
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  fgets(fp);
  fgets(fp);
  char tline[50];
  fgets(tline,50,fp);
  
  int num = sscanf(tline,"<Piece NumberOfPoints=\"%d\"",int nnodesl);
  int nnodesG = nnodesl;
  
  char str1 = "<Piece NumberOfPoints=\"";
  char str2[30];
  snprintf(str2,100,"%d",nnodesl);
  char str3 = "\"";
  strcat(str1,str2);
  strcat(str1,str3);
  char str4 = " NumberOfCells=\"%d\"";
  strcat(str1,str4);
  int num2 = sscanf(tline,str4,int ncellsl);
  int ncellsG = ncellsl;

  nnodes = nnodesG;
  nel = ncellsG;
}

void read_coordinates(char basename, int myrank, size_t nnodes)
{
  hipMallocManaged(&coordinates, (nnodes * 3 * sizeof(unsignedchar)));
  
  int count = 0;
  char str = "<DataArray type=\"Float64\" Name=\"coordinates\" NumberOfComponents=\"3\" format=\"ascii\">";
  int count = 0;
  printf("got here");
  
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++) 
    { 
        // check for alphabets 
        if (isalpha(tline[i]) != 0) 
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
	{
	  break;
	}
      
      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++) 
	{ 
	  // check for alphabets 
	  if (isalpha(tline[i]) != 0) 
	    {
	      alphabet++;
	    }
	}
    }

  for(i=0,i<nnodes,i++)
    {
      fgets(tline,50,fp);
      int num3 = sscanf(tline,"%f %f %f",int x);
      coordinates[count][0] = x[0];
      coordinates[count][1] = x[1];
      coordinates[count][2] = x[2];
      count++;
    }
  fclose(fp);
}

void read_elements(char basename, int myrank, size_t nel)
{
  hipMallocManaged(&coordinates, (nel * 4 * sizeof(unsignedchar)));
  
  int count = 0;
  char str = "<DataArray type=\"Int32\" Name=\"connectivity\" format=\"ascii\">";
  int count = 0;
  printf("got here");
  
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++) 
    { 
        // check for alphabets 
        if (isalpha(tline[i]) != 0) 
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
	{
	  break;
	}
      
      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++) 
	{ 
	  // check for alphabets 
	  if (isalpha(tline[i]) != 0) 
	    {
	      alphabet++;
	    }
	}
    }

  for(i=0,i<nnodes,i++)
    {
      fgets(tline,50,fp);
      int num3 = sscanf(tline,"%f %f %f %f",int x);
      coordinates[count][0] = x[0];
      coordinates[count][1] = x[1];
      coordinates[count][2] = x[2];
      coordinates[count][3] = x[3];
      count++;
    }
  fclose(fp);
}

void read_psi(char basename, int myrank, size_t nel)
{
  hipMallocManaged(&psi, (nel * sizeof(unsignedchar)));
  
  int count = 0;
  char str = "<DataArray type=\"Int32\" Name=\"connectivity\" format=\"ascii\">";
  int count = 0;
  printf("got here");
  
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++) 
    { 
        // check for alphabets 
        if (isalpha(tline[i]) != 0) 
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
	{
	  break;
	}
      
      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++) 
	{ 
	  // check for alphabets 
	  if (isalpha(tline[i]) != 0) 
	    {
	      alphabet++;
	    }
	}
    }

  for(i=0,i<nnodes,i++)
    {
      fgets(tline,50,fp);
      int x = atoi(tline);
      psi[count] = x;
      count++;
    }
  fclose(fp);
}

struct shapeStruct {
  float N, dN, jac;
};

float struct shapeStruct Struct;

Struct shape(float gp, float xe)
{
  //local coordinate
  float r = gp[0];
  float s = gp[1];
  float t = gp[2];

  //Shape functions
  float N = {r, s, t, 1 - r - s - t};
  float N_r = {1, 0, 0, -1};
  float N_s = {0, 1, 0, -1};
  float N_t = {0 ,0 , 1, -1};

  float x_r = N_r[0]*xe[0,0] + N_r[1]*xe[1,0] + N_r[2]*xe[2,0] + N_r[3]*xe[3,0];
  float x_s = N_s[0]*xe[0,0] + N_s[1]*xe[1,0] + N_s[2]*xe[2,0] + N_s[3]*xe[3,0];
  float x_t = N_t[0]*xe[0,0] + N_t[1]*xe[1,0] + N_t[2]*xe[2,0] + N_t[3]*xe[3,0];

  float y_r = N_r[0]*xe[0,1] + N_r[1]*xe[1,1] + N_r[2]*xe[2,1] + N_r[3]*xe[3,1];
  float y_s = N_s[0]*xe[0,1] + N_s[1]*xe[1,1] + N_s[2]*xe[2,1] + N_s[3]*xe[3,1];
  float y_t = N_t[0]*xe[0,1] + N_t[1]*xe[1,1] + N_t[2]*xe[2,1] + N_t[3]*xe[3,1];
  

__global__ void gen_LMArray(unsigned char elements, size_t nel, int ID)
{
  for(i = blockIdx.x * blockDim.x + threadIdx.x; index < nel; index += (blockDim.x * gridDim.x)))
    {
      for(j=0;j<4;j++)
	{
	  LM[j][i] = ID[elements[i][j]];
	}
    }
}

bool gol_runKernel(unsigned char coordinates, size_t nnodes, float powder_thick, float Tol, unsigned char elements,
		   size_t nel, int ID, ushort threadsCount)
{
  //Get boundary nodes
  int count = 0;
  int i;
  int j;
  int k;
  for(i=0;i<nnodes;i++)
    {
      float z = coordinates[i][2];
      if(fabs(z - powder_thick) < Tol)
	{
	  float fixnodes[count] = i;
	  count++;
	}
    }

  //Assembling ID array
  for(i=0;i<nnodes;i++)
    {
      int ID[i] = 1;
    }
  
  int ndispl = sizeof(fixnodes)/sizeof(fixnodes[0]);
  for(i=0;i<ndispl;i++)
    {
      int nd = fixnodes[i];
      Id[nd] = 0;
    }

  //Fill ID array
  count = 0;
  for(j=0;j<nodes;j++)
    {
      if(ID[j] != 0)
	{
	  count++;
	  ID[j] = count;
	}
    }

  //Generate LM array
  size_t reqBlocksCount = ceil(nel/threadsCount); //number of blocks count for the LM array
  unsigned int blocksCount = (unsigned int)min(65536, (unsigned int)reqBlocksCount); // setting blocks count based on the required blocks.
  gen_LMArray<<<blocksCount, threadsCount>>>(elements, nel, ID);

  int ndof = 0;
  float d;
  for(i = 0;i < nnodes;i++)
    {
      //Displacement Vector
      d[i] = 0;
      if(ID[i] > ndof)
	{
	  ndof = ID[i];
	}
    }
  
  //Compute Sparcity
  nzmax = 0;
  int elem;
  for(elem = 0;elem < nel;elem++)
    {
      for(k = 0;k < 4;k++)
	{
	  int i_index = LM[k][elem];
	  if(i_index > 0)
	    {
	      for(m = 0;m < 4;m++)
		{
		  int j_index = LM[m][elem];
		  if(j_index > 0)
		    {
		      nzmax++;
		    }
		}
	    }
	}
    }

  for(i = 0;i < nzmax;i++)
    {
      irow[i] = 0;
      icol[i] = 0;
    }
  
  count = 0;
  for(elem = 0;elem < nel;elem++)
    {
      for(k = 0;k < 4;k++)
	{
	  i_index = LM[k][elem];
	  if(i_index > 0)
	    {
	      for(m = 0;m < 4;m++)
		{
		  j_index = LM[m][elem];
		  if(j_index > 0)
		    {
		      irow[count] = i_index;
		      icol[count] = j_index;
		      count++;
		    }
		}
	    }
	}
    }

  //Assembling stiffness matrix
  float K;
  float F;
  for(i = 0;i < nzmax;i++)
    {
      K[i] = 0;
    }
  for(i = 0;i < ndof;i++)
    {
      F[i] = 0;
    }
  for(i = 0; i < nel;i++)
    {
      float xe[4][3] = {
			{coordinates[elements[i][0]][0], coordinates[elements[i][0]][1], coordinates[elements[i][0]][2]},
			{coordinates[elements[i][1]][0], coordinates[elements[i][1]][1], coordinates[elements[i][1]][2]},
			{coordinates[elements[i][2]][0], coordinates[elements[i][2]][1], coordinates[elements[i][2]][2]},
			{coordinates[elements[i][3]][0], coordinates[elements[i][3]][1], coordinates[elements[i][3]][2]}
      };
      float Psie = psi[i];
      
  
}
