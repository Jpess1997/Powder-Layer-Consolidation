#include "hip/hip_runtime.h"
//Secondary function code to run on the GPU with cuda
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <stdlib.h>
#include <math.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

extern unsigned char *coordinates;

extern unsigned char *elements;

extern size_t nel, nnodes, np, nzmax;

extern char basename;

extern unsigned char *psi;

extern unsigned char *LM;

extern unsigned char *irow;

extern unsigned char *icol;

extern "C"
{
  extern void num_ElementsNodes(char basename, int myrank);

  extern void read_coordinates(char basename, int myrank, size_t nnodes);

  extern void read_elements(char basename, int myrank, size_t nel);

  extern void read_psi(char basename, int myrank, size_t nel);

  extern bool gol_runKernel(unsigned char coordinates, size_t nnodes, float powder_thick, float Tol, unsigned char elements,
			  size_t nel, ushort threadsCount);
}

void num_ElementsNodes(char basename, int myrank)
{
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  fgets(fp);
  fgets(fp);
  char tline[50];
  fgets(tline,50,fp);
  
  int num = sscanf(tline,"<Piece NumberOfPoints=\"%d\"",int nnodesl);
  int nnodesG = nnodesl;
  
  char str1 = "<Piece NumberOfPoints=\"";
  char str2[30];
  snprintf(str2,100,"%d",nnodesl);
  char str3 = "\"";
  strcat(str1,str2);
  strcat(str1,str3);
  char str4 = " NumberOfCells=\"%d\"";
  strcat(str1,str4);
  int num2 = sscanf(tline,str4,int ncellsl);
  int ncellsG = ncellsl;

  nnodes = nnodesG;
  nel = ncellsG;
}

void read_coordinates(char basename, int myrank, size_t nnodes)
{
  //hipMallocManaged(&coordinates, (nnodes * 3 * sizeof(unsignedchar)));
  
  int count = 0;
  char str = "<DataArray type=\"Float64\" Name=\"coordinates\" NumberOfComponents=\"3\" format=\"ascii\">";
  int count = 0;
  printf("got here");
  
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++) 
    { 
        // check for alphabets 
        if (isalpha(tline[i]) != 0) 
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
	{
	  break;
	}
      
      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++) 
	{ 
	  // check for alphabets 
	  if (isalpha(tline[i]) != 0) 
	    {
	      alphabet++;
	    }
	}
    }

  for(i=0,i<nnodes,i++)
    {
      fgets(tline,50,fp);
      int num3 = sscanf(tline,"%f %f %f",int x);
      coordinates[count][0] = x[0];
      coordinates[count][1] = x[1];
      coordinates[count][2] = x[2];
      count++;
    }
  fclose(fp);
}

void read_elements(char basename, int myrank, size_t nel)
{
  //hipMallocManaged(&coordinates, (nel * 4 * sizeof(unsignedchar)));
  
  int count = 0;
  char str = "<DataArray type=\"Int32\" Name=\"connectivity\" format=\"ascii\">";
  int count = 0;
  printf("got here");
  
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++) 
    { 
        // check for alphabets 
        if (isalpha(tline[i]) != 0) 
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
	{
	  break;
	}
      
      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++) 
	{ 
	  // check for alphabets 
	  if (isalpha(tline[i]) != 0) 
	    {
	      alphabet++;
	    }
	}
    }

  for(i=0,i<nnodes,i++)
    {
      fgets(tline,50,fp);
      int num3 = sscanf(tline,"%f %f %f %f",int x);
      coordinates[count][0] = x[0];
      coordinates[count][1] = x[1];
      coordinates[count][2] = x[2];
      coordinates[count][3] = x[3];
      count++;
    }
  fclose(fp);
}

void read_psi(char basename, int myrank, size_t nel)
{
  //hipMallocManaged(&psi, (nel * sizeof(unsignedchar)));
  
  int count = 0;
  char str = "<DataArray type=\"Int32\" Name=\"connectivity\" format=\"ascii\">";
  int count = 0;
  printf("got here");
  
  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(basename,fname);
  FILE *fp;
  fp = fopen(basename,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++) 
    { 
        // check for alphabets 
        if (isalpha(tline[i]) != 0) 
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
	{
	  break;
	}
      
      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++) 
	{ 
	  // check for alphabets 
	  if (isalpha(tline[i]) != 0) 
	    {
	      alphabet++;
	    }
	}
    }

  for(i=0,i<nnodes,i++)
    {
      fgets(tline,50,fp);
      int x = atoi(tline);
      psi[count] = x;
      count++;
    }
  fclose(fp);
}

struct shapeStruct {
  float N, dN, jac;
};

float struct shapeStruct Struct;

Struct shape(float gp, float xe)
{
  Struct s;
  int i;
  //local coordinate
  float r = gp[0];
  float s = gp[1];
  float t = gp[2];

  //Shape functions
  float N = {r, s, t, 1 - r - s - t};
  float N_r = {1, 0, 0, -1};
  float N_s = {0, 1, 0, -1};
  float N_t = {0 ,0 , 1, -1};

  float x_r = N_r[0]*xe[0][0] + N_r[1]*xe[1][0] + N_r[2]*xe[2][0] + N_r[3]*xe[3][0];
  float x_s = N_s[0]*xe[0][0] + N_s[1]*xe[1][0] + N_s[2]*xe[2][0] + N_s[3]*xe[3][0];
  float x_t = N_t[0]*xe[0][0] + N_t[1]*xe[1][0] + N_t[2]*xe[2][0] + N_t[3]*xe[3][0];

  float y_r = N_r[0]*xe[0][1] + N_r[1]*xe[1][1] + N_r[2]*xe[2][1] + N_r[3]*xe[3][1];
  float y_s = N_s[0]*xe[0][1] + N_s[1]*xe[1][1] + N_s[2]*xe[2][1] + N_s[3]*xe[3][1];
  float y_t = N_t[0]*xe[0][1] + N_t[1]*xe[1][1] + N_t[2]*xe[2][1] + N_t[3]*xe[3][1];

  float z_r = N_r[0]*xe[0][2] + N_r[1]*xe[1][2] + N_r[2]*xe[2][2] + N_r[3]*xe[3][2];
  float z_s = N_s[0]*xe[0][2] + N_s[1]*xe[1][2] + N_s[2]*xe[2][2] + N_s[3]*xe[3][2];
  float z_t = N_t[0]*xe[0][2] + N_t[1]*xe[1][2] + N_t[2]*xe[2][2] + N_t[3]*xe[3][2];

  float jacobian[3][3] = {x_r,x_s,x_t,y_r,y_s,y_t,z_r,z_s,z_t};
  float jacDet = x_r*(y_s*z_t - y_t*z_s) - x_s*(y_r*z_t - t_t*z_r) + x_t*(y_r*z_s - y_s*z_r);
  float jac = abs(jacDet);

  //Check Jacobian
  if(jac <= 0.0)
    {
      fprintf(strerr,"Negative jacobian, element too distorted!\n");
    }
  //Take the inverse of the Jacobian
  float inv_jac[3][3] = {(y_s*z_t - y_t*z_s)/jacDet, (x_t*z_s - x_s*z_t)/jacDet, (x_s*y_t - x_t*y_s)/jacDet,
			 (y_t*z_r - y_r*z_t)/jacDet, (x_r*z_t - x_t*z_r)/jacDet, (x_t*y_r - x_r*y_t)/jacDet,
			 (y_r*z_s - y_s*z_r)/jacDet, (x_s*z_r - x_r*z_s)/jacDet, (x_r*y_s - x_s*y_r)/jacDet};
  float dN;
  for(i=0;i<4,i++)
    {
      dN[i][0] = N_r[i]*inv_jac[0][0] + N_s[i]*inv_jac[1][0] + N_t[i]*inv_jac[2][0];
      dN[i][1] = N_r[i]*inv_jac[0][1] + N_s[i]*inv_jac[1][1] + N_t[i]*inv_jac[2][1];
      dN[i][2] = N_r[i]*inv_jac[0][2] + N_s[i]*inv_jac[1][2] + N_t[i]*inv_jac[2][2];
    }
  s.N = N;
  s.dN = dN;
  s.jac = jac;

  return s;
}

struct weakformStruct {
  float ke, fe;
};

float weakformStruct Struct;

Struct weakform(float xe, float Psie, float porosity)
{
  int i, j, k;
  // 1 point formula - degree of precision 1
  float gp = {0.25, 0.25, 0.25};
  int w = 1;

  int ngp = 1;

  //initialize stiffness matrix
  float ke[4][4] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  //right hand size
  float fe[4][1] = {0,0,0,0};

  //stress strain displacement matrix
  float B[1][4] = {0,0,0,0};
  //loop over gauss points
  Struct result = shape(gp,xe);
  float N = result.N;
  float dN = result.dN;
  float jac = result.jac;
  for(i=0;i<ngp;i++)
    {
      float por = porosity;
      float z = N * {xe[0][2], xe[1][2], xe[2][2], xe[3][2]};
      if( z < powderThick)
	{
	  por = 0.0;
	}
      for(j=0;j<4;j++)
	{
	  B[j] = dN[j][3];
	}
      //Transpose of N
      float Ntr[4][1] = {N[0],N[1],N[2],N[3]};
      //fill k
      ke = ke + Ntr * B * w[i] * jac;
      //fill fe
      fe = fe - Ntr * ((por * Psie)/(1 - por * (1 - Psie)))*w[i]*jac;
    }
  s.ke = ke;
  s.fe = fe;

  return s;
}


__global__ void gen_LMArray(unsigned char elements, size_t nel, int ID)
{
  for(i = blockIdx.x * blockDim.x + threadIdx.x; i < nel; i += (blockDim.x * gridDim.x))
    {
      for(j=0;j<4;j++)
	{
	  LM[j][i] = ID[elements[i][j]];
	}
    }
}

__global__ float gen_corrector(size_t nnodes, int ID, float a_bar, float powderThick)
{
  int i, index;
  float d;
  for(i = blockIdx.x * blockDim.x + threadIdx.x; i < nnodes; i += (blockDim.x * gridDim.x))
    {
      index = ID[i];
      if(index != 0)
	{
	  d[i] = a_bar[index];
	  if(d[i] > powderThick)
	    {
	      d[i] = 0;
	    }
	  if(isdigit(round(d[i])) == 0)
	    {
	      d[i] = 0;
	    }
	}
    }

  return d
}

bool gol_runKernel(unsigned char coordinates, size_t nnodes, float powder_thick, float Tol, unsigned char elements,
		   size_t nel, ushort threadsCount)
{
  //Get boundary nodes
  int count = 0;
  int i;
  int j;
  int k;
  for(i=0;i<nnodes;i++)
    {
      float z = coordinates[i][2];
      if(fabs(z - powder_thick) < Tol)
	{
	  float fixnodes[count] = i;
	  count++;
	}
    }

  hipMallocManaged(&ID, (nnodes * sizeof(int)));
  
  //Assembling ID array
  int ID;
  for(i=0;i<nnodes;i++)
    {
      ID[i] = 1;
    }
  
  int ndispl = sizeof(fixnodes)/sizeof(fixnodes[0]);
  int nd;
  for(i=0;i<ndispl;i++)
    {
      nd = fixnodes[i];
      Id[nd] = 0;
    }

  //Fill ID array
  count = 0;
  for(j=0;j<nodes;j++)
    {
      if(ID[j] != 0)
	{
	  count++;
	  ID[j] = count;
	}
    }
  
  hipMallocManaged(&LM, (nel * sizeof(unsigned char)));
  
  //Generate LM array
  size_t reqBlocksCount = ceil(nel/threadsCount); //number of blocks count for the LM array
  unsigned int blocksCount = (unsigned int)min(65536, (unsigned int)reqBlocksCount); // setting blocks count based on the required blocks.
  gen_LMArray<<<blocksCount, threadsCount>>>(elements, nel, ID);
  hipDeviceSynchronize();
  
  int ndof = 0;
  float d;
  for(i = 0;i < nnodes;i++)
    {
      //Displacement Vector
      d[i] = 0;
      if(ID[i] > ndof)
	{
	  ndof = ID[i];
	}
    }
  
  //Compute Sparcity
  nzmax = 0;
  int elem, i_index, j_index;
  for(elem = 0;elem < nel;elem++)
    {
      for(k = 0;k < 4;k++)
	{
	  i_index = LM[k][elem];
	  if(i_index > 0)
	    {
	      for(m = 0;m < 4;m++)
		{
		  j_index = LM[m][elem];
		  if(j_index > 0)
		    {
		      nzmax++;
		    }
		}
	    }
	}
    }

  for(i = 0;i < nzmax;i++)
    {
      irow[i] = 0;
      icol[i] = 0;
    }
  
  count = 0;
  for(elem = 0;elem < nel;elem++)
    {
      for(k = 0;k < 4;k++)
	{
	  i_index = LM[k][elem];
	  if(i_index > 0)
	    {
	      for(m = 0;m < 4;m++)
		{
		  j_index = LM[m][elem];
		  if(j_index > 0)
		    {
		      irow[count] = i_index;
		      icol[count] = j_index;
		      count++;
		    }
		}
	    }
	}
    }

  //Assembling stiffness matrix
  float K;
  float F;
  float xe;
  float Psie;
  Struct result;
  float ke;
  float fe;
  count = 0;
  for(i = 0;i < nzmax;i++)
    {
      K[i] = 0;
    }
  for(i = 0;i < ndof;i++)
    {
      F[i][0] = 0;
    }
  for(i = 0; i < nel;i++)
    {
      xe[4][3] = {
		  {coordinates[elements[i][0]][0], coordinates[elements[i][0]][1], coordinates[elements[i][0]][2]},
		  {coordinates[elements[i][1]][0], coordinates[elements[i][1]][1], coordinates[elements[i][1]][2]},
		  {coordinates[elements[i][2]][0], coordinates[elements[i][2]][1], coordinates[elements[i][2]][2]},
		  {coordinates[elements[i][3]][0], coordinates[elements[i][3]][1], coordinates[elements[i][3]][2]}
      };
      Psie = psi[i];
      result = weakform(xe,Psie,porosity);
      ke = result.ke;
      fe = result.fe;
      for(j=0;j<4;j++)
	{
	  i_index = LM[j][i];
	  if(i_index > 0)
	    {
	      F[i_index] = F[i_index] + fe[j];
	      for(k=0;k<4;k++)
		{
		  j_index = LM[k][i];
		  if(j_index > 0)
		    {
		      K[count] = K[count] + ke[j][k];
		    }
		}
	    }
	}
    }

  float M;
  //Ensamble sparse matrix
  for(i=0;i<ndof;i++)
    {
      for(j=0;j<ndof;j++)
	{
	  M[i][j] = 0;
	}
    }
  for(i=0;i<nzmax;i++)
    {
      M[irow[i]][icol[i]] = M[irow[i]][icol[i]] + K[i];
    }
  //For future work implement solver for solving system of equation M * a_bar = F
  hipMallocManaged(&a_bar, (ndof * sizeof(float)));
  
  float a_bar;
  a_bar = F;

  //Change small values to zero
  for(i=0;i<ndof;i++)
    {
      if(a_bar[i] < 0.0000001)
	{
	  a_bar[i] = 0;
	}
    }

  //Corrector phase to be done in cuda
  reqBlocksCount = ceil(nnodes/threadsCount); //number of blocks count for the LM array
  blocksCount = (unsigned int)min(65536, (unsigned int)reqBlocksCount); // setting blocks count based on the required blocks.
  d = gen_corrector<<<blocksCount, threadsCount>>>(nnodes, ID, a_bar, powderThick);
  hipDeviceSynchronize();
  
  for(i=0;i<nnodes;i++)
    {
      coordinates[i][2] = coordinates[i][2] + d[i];
    }
}
