#include "hip/hip_runtime.h"
//Secondary function code to run on the GPU with cuda
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <stdlib.h>
#include <math.h>
#include<unistd.h>
#include<stdbool.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

float nnodesl;

extern int *ID;

extern float *coordinates;

extern float *elements;

extern int nel, nnodes, np, nzmax;

extern char baseName[80];

extern float *psi;

extern float *LM;

extern int *irow;

extern int *icol;

extern float powderThick;

float *N;

float *dN;

float jac;

float *ke;

float *fe;

extern float *d;

extern float *a_bar;

extern "C"
{
  extern void num_ElementsNodes(char baseName [80], int myrank);

  extern int offsetCalc(char baseName [80], int numranks, int myrank);

  extern void read_coordinates(char baseName [80], int myrank, int nnodes);

  extern void read_elements(char baseName [80], int myrank, int nel);

  extern void read_psi(char baseName [80], int myrank, int nel);

  extern bool gol_runKernel(float *coordinates, int nnodes, float powderThick,
			    float Tol, float *elements,
			    int nel, int **d_ID, ushort threadsCount,
			    float **d_d, float **d_a_bar);

  extern void gol_freeData();
}

void printLine(int line)
{
  char fileName[30];
  snprintf(fileName,100,"errorAtLineCuda.txt");
  FILE *fp;
  fp = fopen(fileName,"w+");
  fprintf(fp,"Line is %d.\n",line);
  fclose(fp);
}

void num_ElementsNodes(char baseName [80], int myrank)
{
  printLine(__LINE__);
  char fname [100];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(baseName, fname);
  FILE *fp;
  fp = fopen(baseName,"r");
  printLine(__LINE__);
  
  fgets(baseName, 47, fp);
  char tline[50];
  fgets(tline, 50, fp);
  printLine(__LINE__);
  
  int num = sscanf(tline,"<Piece NumberOfPoints=\"%d\"",&nnodesl);
  int nnodesG = nnodesl;
  printLine(__LINE__);
  
  char str1 [24] = "<Piece NumberOfPoints=\"";
  char str2 [30];
  snprintf(str2,100,"%d",nnodesl);
  char str3 [3] = "\"";
  strcat(str1,str2);
  strcat(str1,str3);
  char str4 [100] = "NumberOfCells=\"%d\"";
  strcat(str1,str4);
  int ncellsl;
  int num2 = sscanf(tline,str4, &ncellsl);
  int ncellsG = ncellsl;
  printLine(__LINE__);
  
  nnodes = nnodesG;
  nel = ncellsG;
}

int offsetCalc(char baseName [80], int numranks, int myrank)
{
  int i;
  char fname[30];
  FILE *fp;
  char tline[50];
  float nnodesG[numranks];
  float offset[numranks];
  for(i=0;i<numranks;i++)
    {
      snprintf(fname,100,"%u.vtu",myrank);
      strcat(baseName,fname);
      fp = fopen(baseName,"r");

      fgets(tline,50,fp);

      float num = sscanf(tline,"<Piece NumberOfPoints=\"%d\"", nnodesl);
      nnodesG[i] = nnodesl;
    }

  for(i=0;i<numranks;i++)
    {
      offset[i] = 0;
    }

  for(i=1;i<numranks;i++)
    {
      offset[i] = offset[i-1] + nnodesG[i-1];
    }
    return offset[numranks];
}

void read_coordinates(char baseName [80], int myrank, int nnodes)
{
  int a,b;
  //float coordinates[nnodes][3];
  hipMallocManaged(&coordinates, ((nnodes * 3) * sizeof(float)));
  //size_t pitch;
  //hipMallocPitch(&coordinates, &pitch, sizeof(float)*3, nnodes);
  for (a=0; a<(nnodes); a++)
    {
      for(b=0;b<3;b++)
	{
	  coordinates[a+b*nnodes] = 0;
	}
    }


  int count = 0;
  char str[100] = "<DataArray type=\"Float64\" Name=\"coordinates\" NumberOfComponents=\"3\" format=\"ascii\">";
  printf("got here");

  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(baseName,fname);
  FILE *fp;
  fp = fopen(baseName,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++)
    {
        // check for alphabets 
        if (isalpha(tline[i]) != 0)
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
        {
          break;
        }

      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++)
        {
          // check for alphabets 
          if (isalpha(tline[i]) != 0)
            {
              alphabet++;
            }
        }
    }
  //coordinates[nnodes][3];
  for(i=0;i<nnodes;i++)
    {
      float x,y,z;
      //float coordinates[nnodes][3];
      fgets(tline,50,fp);
      int num3 = sscanf(tline,"%f %f %f", &x,&y,&z);
      coordinates[count] = x;
      coordinates[count+nnodes] = y;
      coordinates[count+2*nnodes] = z;
      count++;
    }
  fclose(fp);
}

void read_elements(char baseName [80], int myrank, int nel)
{
  hipMallocManaged(&elements, ((nel * 4) * sizeof(float)));
  int a;
  for (a=0; a<(4*nel); a++)
    {
      elements[a] = 0;
    }

  int count = 0;
  char str[80] = "<DataArray type=\"Int32\" Name=\"connectivity\" format=\"ascii\">";
  printf("got here");

  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(baseName,fname);
  FILE *fp;
  fp = fopen(baseName,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++)
    {
        // check for alphabets 
        if (isalpha(tline[i]) != 0)
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
        {
          break;
        }

      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++)
        {
          // check for alphabets 
          if (isalpha(tline[i]) != 0)
            {
              alphabet++;
            }
        }
    }

  for(i=0;i<nel;i++)
    {
      fgets(tline,50,fp);
      float x,y,z,w;
      //float elements[nel][4];
      int num3 = sscanf(tline,"%f %f %f %f", &x,&y,&z,&w);
      elements[count] = x;
      elements[count+nel] = y;
      elements[count+2*nel] = z;
      elements[count+3*nel] = w;
      count++;
    }
  fclose(fp);
}

void read_psi(char baseName [80], int myrank, int nel)
{
  hipMallocManaged(&psi, (nel * sizeof(float)));

  int count = 0;
  char str[100] = "<DataArray type=\"Int32\" Name=\"connectivity\" format=\"ascii\">";
  printf("got here");

  char fname[30];
  snprintf(fname,100,"%u.vtu",myrank);
  strcat(baseName,fname);
  FILE *fp;
  fp = fopen(baseName,"r");

  char tline[50];
  fgets(tline,50,fp);

  int i;
  int alphabet = 0;
  for (i=0; tline[i]!= '\0'; i++)
    {
        // check for alphabets 
        if (isalpha(tline[i]) != 0)
        {
            alphabet++;
        }
    }
  while(alphabet > 0)
    {
      if(strcmp(tline,str) == 0)
        {
          break;
        }

      fgets(tline,50,fp);

      alphabet = 0;
      for (i=0; tline[i]!= '\0'; i++)
        {
          // check for alphabets 
          if (isalpha(tline[i]) != 0)
            {
              alphabet++;
            }
        }
    }

  for(i=0;i<nel;i++)
    {
      fgets(tline,50,fp);
      float x = atoi(tline);
      psi[count] = x;
      count++;
    }
  fclose(fp);
}

//struct shapeStruct {
//  float NS, dNS, jacS;
//};

//typedef struct shapeStruct Struct;

//Struct shape(float gp[3], float xe[4][3])
//void shape(float gp[3], float xe[4][3], float *add_N[], float *add_dN[][], float *add_jac)
void shape(float gp[3], float xe[12])
{
  //struct s;
  int i;
  //float xe[4][3];
  //float gp[3];
  //local coordinate
  float r = gp[0];
  float s = gp[1];
  float t = gp[2];

  //Shape functions
  //N[4] = {r, s, t, 1 - r - s - t};
  N[0] = r;
  N[1] = s;
  N[2] = t;
  N[3] = 1-r-s-t;
  float N_r[4] = {1, 0, 0, -1};
  float N_s[4] = {0, 1, 0, -1};
  float N_t[4] = {0 ,0 , 1, -1};

  float x_r = N_r[0]*xe[0+0*4] + N_r[1]*xe[1+0*4] + N_r[2]*xe[2+0*4] + N_r[3]*xe[3+0*4];
  float x_s = N_s[0]*xe[0+0*4] + N_s[1]*xe[1+0*4] + N_s[2]*xe[2+0*4] + N_s[3]*xe[3+0*4];
  float x_t = N_t[0]*xe[0+0*4] + N_t[1]*xe[1+0*4] + N_t[2]*xe[2+0*4] + N_t[3]*xe[3+0*4];

  float y_r = N_r[0]*xe[0+1*4] + N_r[1]*xe[1+1*4] + N_r[2]*xe[2+1*4] + N_r[3]*xe[3+1*4];
  float y_s = N_s[0]*xe[0+1*4] + N_s[1]*xe[1+1*4] + N_s[2]*xe[2+1*4] + N_s[3]*xe[3+1*4];
  float y_t = N_t[0]*xe[0+1*4] + N_t[1]*xe[1+1*4] + N_t[2]*xe[2+1*4] + N_t[3]*xe[3+1*4];

  float z_r = N_r[0]*xe[0+2*4] + N_r[1]*xe[1+2*4] + N_r[2]*xe[2+2*4] + N_r[3]*xe[3+2*4];
  float z_s = N_s[0]*xe[0+2*4] + N_s[1]*xe[1+2*4] + N_s[2]*xe[2+2*4] + N_s[3]*xe[3+2*4];
  float z_t = N_t[0]*xe[0+2*4] + N_t[1]*xe[1+2*4] + N_t[2]*xe[2+2*4] + N_t[3]*xe[3+2*4];

  float jacDet = x_r*(y_s*z_t - y_t*z_s) - x_s*(y_r*z_t - y_t*z_r) + x_t*(y_r*z_s - y_s*z_r);
  jac = abs(jacDet);

  //Check Jacobian
  //if(jac =< 0.0)
  //  {
  //    fprintf(stderr, "Negative jacobian, element too distorted!\n");
  //  }
  //Take the inverse of the Jacobian
  float inv_jac[3][3] = {(y_s*z_t - y_t*z_s)/jacDet, (x_t*z_s - x_s*z_t)/jacDet, (x_s*y_t - x_t*y_s)/jacDet,
                         (y_t*z_r - y_r*z_t)/jacDet, (x_r*z_t - x_t*z_r)/jacDet, (x_t*y_r - x_r*y_t)/jacDet,
                         (y_r*z_s - y_s*z_r)/jacDet, (x_s*z_r - x_r*z_s)/jacDet, (x_r*y_s - x_s*y_r)/jacDet};
  //dN[4][3];
  for(i=0;i<4;i++)
    {
      dN[i+0*4] = N_r[i]*inv_jac[0][0] + N_s[i]*inv_jac[1][0] + N_t[i]*inv_jac[2][0];
      dN[i+1*4] = N_r[i]*inv_jac[0][1] + N_s[i]*inv_jac[1][1] + N_t[i]*inv_jac[2][1];
      dN[i+2*4] = N_r[i]*inv_jac[0][2] + N_s[i]*inv_jac[1][2] + N_t[i]*inv_jac[2][2];
    }
  //s.NS = N;
  //s.dNS = dN;
  //s.jacS = jac;

  //*add_N = N[];
  //*add_dN = dN[][];
  //*add_jac = jac;

  //return s;
}

//struct weakformStruct {
//  float ke, fe;
//};

//typedef struct weakformStruct Struct1;

//Struct1 weakform(float xe[4][3], float Psie, float porosity);
void weakform(float xe[12], float Psie, float porosity)
{
  int i;
  int j;
  int k;
  int l;
  //float xe[4][3];
  //float N, dN, jac;
  // 1 point formula - degree of precision 1
  float gp[3] = {0.25, 0.25, 0.25};
  int w = 1;

  int ngp = 1;

  //initialize stiffness matrix
  //float ke[4][4] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  for(i=0;i<16;i++)
    {
      ke[i] = 0;
    }
  
  //right hand size
  //float fe[4][1] = {0,0,0,0};
  for(i=0;i<4;i++)
    {
      fe[i] = 0;
    }

  //stress strain displacement matrix
  float B[4] = {0,0,0,0};
  //loop over gauss points
  //struct result = shape(gp, xe[4][3]);
  //shape(gp,xe,&N,&dN,&jac);
  shape(gp,xe);
  //float N = result.N;
  //float dN = result.dN;
  //float jac = result.jac;
  for(i=0;i<ngp;i++)
    {
      float por = porosity;
      //float z = N * {xe[0][2], xe[1][2], xe[2][2], xe[3][2]};
      float z = N[0]*xe[0+2*4] + N[1]*xe[1+2*4] + N[2]*xe[2+2*4] + N[3]*xe[3+2*4];
      if( z < powderThick)
        {
          por = 0.0;
        }
      for(j=0;j<4;j++)
        {
          B[j] = dN[j*2*4];
        }
      //Transpose of N
      float Ntr[4] = {N[0],N[1],N[2],N[3]};
      //fill k
      for(k=0;k<4;k++)
	{
	  for(l=0;l<4;l++)
	    {
	      ke[k+1] = ke[k+l] + Ntr[l] * B[k] * w * jac;
	    }
	}
      //fill fe
      for(k=0;k<4;k++)
	{
	  fe[k] = fe[k] - Ntr[k] * ((por * Psie)/(1 - por * (1 - Psie)))*w*jac;
	}
    }
  //float s.ke = ke;
  //float s.fe = fe;

  //return s;
}

/*
__global__ void gen_LMArray(float elements, int nel)
{
  int i,j;
  for(i = blockIdx.x * blockDim.x + threadIdx.x; i < nel; i += (blockDim.x * gridDim.x))
    {
      for(j=0;j<4;j++)
        {
          LM[j+i*nel] = ID[elements[i+j*nel]];
        }
    }
}
*/

__global__ void gen_corrector(int nnodes, int *d_ID, float *d_a_bar, float powderThick, float *d_d)
{
  int i, index;
  //float d;
  for(i = blockIdx.x * blockDim.x + threadIdx.x; i < nnodes; i += (blockDim.x * gridDim.x))
    {
      index = d_ID[i];
      if(index != 0)
        {
          d_d[i] = d_a_bar[index];
          if(d_d[i] > powderThick)
            {
              d_d[i] = 0;
            }
          //if(isdigit(round(d_d[i])) == 0)
          //  {
          //    d_d[i] = 0;
          //  }
        }
    }

  //return d
}

bool gol_runKernel(float *coordinates, int nnodes, float powderThick,float Tol, float *elements, int nel, int **d_ID, ushort threadsCount, float **d_d, float **d_a_bar)
{
  //Get boundary nodes
  int count = 0;
  int i;
  int j;
  //int k;
  //int m;
  int *fixnodes;
  //float *xe;
  for(i=0;i<nnodes;i++)
    {
      float z = coordinates[i+2*nnodes];
      if(fabs(z - powderThick) < Tol)
        {
          fixnodes[count] = i;
          count++;
        }
    }

  hipMallocManaged(&ID, (nnodes * sizeof(int)));
  hipMallocManaged(&d, (nnodes * sizeof(float)));

  //Assembling ID array
  float ID[nnodes];
  for(i=0;i<nnodes;i++)
    {
      ID[i] = 1;
    }

  int ndispl = sizeof(fixnodes)/sizeof(fixnodes[0]);
  int nd;
  int g;
  for(g=0; g<ndispl; g++)
    {
      nd = fixnodes[g];
      ID[nd] = 0;
    }

  //Fill ID array
  count = 0;
  for(j=0;j<nnodes;j++)
    {
      if(ID[j] != 0)
        {
          count++;
          ID[j] = count;
        }
    }

  int ndof = 0;
  //float d;
  for(i = 0;i < nnodes;i++)
    {
      //Displacement Vector
      d[i] = 0;
      if(ID[i] > ndof)
        {
          ndof = ID[i];
        }
    }
  
  /*
  hipMallocManaged(&LM, (nel * sizeof(unsigned char)));

  //Generate LM array
  size_t reqBlocksCount = ceil(nel/threadsCount); //number of blocks count for the LM array
  unsigned int blocksCount = (unsigned int)min(65536, (unsigned int)reqBlocksCount); // setting blocks count based on the required blocks.
  int gen_LMArray<<<blocksCount, threadsCount>>>(elements, nel, ID);
  hipDeviceSynchronize();

  //Compute Sparcity
  nzmax = 0;
  int elem, i_index, j_index;
  for(elem = 0;elem < nel;elem++)
    {
      for(k = 0;k < 4;k++)
        {
          i_index = LM[k+elem*nel];
          if(i_index > 0)
            {
              for(m = 0;m < 4;m++)
                {
                  j_index = LM[m+elem*nel];
                  if(j_index > 0)
                    {
                      nzmax++;
                    }
                }
            }
        }
    }

  for(i = 0;i < nzmax;i++)
    {
      irow[i] = 0;
      icol[i] = 0;
    }

  count = 0;
  for(elem = 0; elem < nel; elem++)
    {
      for(k = 0;k < 4;k++)
        {
          i_index = LM[k][elem];
          if(i_index > 0)
            {
              for(m = 0;m < 4;m++)
                {
                  j_index = LM[m][elem];
                  if(j_index > 0)
                    {
                      irow[count] = i_index;
                      icol[count] = j_index;
                      count++;
                    }
                }
            }
        }
    }

  //Assembling stiffness matrix
  float K;
  float F[ndof];
  float Psie;
  struct result;
  count = 0;
  for(i = 0;i < nzmax;i++)
    {
      K[i] = 0;
    }
  for(i = 0;i < ndof;i++)
    {
      F[i] = 0;
    }
  for(i = 0; i < nel;i++)
    {
      for(j=0;j<3;j++)
	{
	  //xe[4][3] = {
	  //	      {coordinates[elements[i][0]][0], coordinates[elements[i][0]][1], coordinates[elements[i][0]][2]},
	  //	      {coordinates[elements[i][1]][0], coordinates[elements[i][1]][1], coordinates[elements[i][1]][2]},
	  //	      {coordinates[elements[i][2]][0], coordinates[elements[i][2]][1], coordinates[elements[i][2]][2]},
	  //	      {coordinates[elements[i][3]][0], coordinates[elements[i][3]][1], coordinates[elements[i][3]][2]}
	  //};
	  xe[0+j*4] = coordinates[elements[i+0*nel]+j*nnodes];
	  xe[1+j*4] = coordinates[elements[i+1*nel]+j*nnodes];
	  xe[2+j*4] = coordinates[elements[i+2*nel]+j*nnodes];
	  xe[3+j*4] = coordinates[elements[i+3*nel]+j*nnodes];
	}
      Psie = psi[i];
      result = weakform(xe,Psie,porosity);
      //ke = result.ke;
      //fe = result.fe;
      for(j=0;j<4;j++)
        {
          i_index = LM[j+i*nel];
          if(i_index > 0)
            {
              F[i_index] = F[i_index] + fe[j];
              for(k=0;k<4;k++)
                {
                  j_index = LM[k][i];
                  if(j_index > 0)
                    {
                      K[count] = K[count] + ke[j+k*4];
                    }
                }
            }
        }
    }

  float M;
  //Ensamble sparse matrix
  for(i=0;i<ndof;i++)
    {
      for(j=0;j<ndof;j++)
        {
          M[i+j*ndof] = 0;
        }
    }
  for(i=0;i<nzmax;i++)
    {
      M[irow[i]+icol[i]*nzmax] = M[irow[i]+icol[i]*nzmax] + K[i];
    }

  */
  //For future work implement solver for solving system of equation M * a_bar = F
  hipMallocManaged(&a_bar, (ndof * sizeof(float)));

  float *F;
  for(i = 0;i < ndof;i++)
    {
      F[i] = 0;
    }
  
  float *a_bar;

  for(i=0; i<ndof;i++)
  {
      a_bar[i] = F[i];
  }

  //Change small values to zero
  for(i=0;i<ndof;i++)
    {
      if(a_bar[i] < 0.0000001)
        {
          a_bar[i] = 0;
        }
    }

  //Corrector phase to be done in cuda 
  size_t reqBlocksCount2 = ceil(nnodes/threadsCount); //number of blocks count for the LM array
  unsigned int blocksCount2 = (unsigned int)min(65536, (unsigned int)reqBlocksCount2);
  gen_corrector<<<blocksCount2, threadsCount>>>(nnodes, *d_ID, *d_a_bar, powderThick, *d_d);
  hipDeviceSynchronize();

  for(i=0;i<nnodes;i++)
    {
      coordinates[i+2*nnodes] = coordinates[i+2*nnodes] + d[i];
    }

  return 0;
}

//free the cuda data that is allocated.
void gol_freeData()
{
 hipFree(coordinates);
 hipFree(elements);
 hipFree(psi);
 hipFree(d);
 hipFree(ID);
 hipFree(a_bar);
}
